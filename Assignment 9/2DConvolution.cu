
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <time.h>
#include <stdlib.h>
const int n = 1024;
const int maskdim = 7;

// CPU function to calculate 2D convolution
void calculate2DConvolutionCPU(int *image, int *mask, int *result, int n, int maskdim) {
    int offset = maskdim / 2;
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
        result[row * n + col] = 0;
        for (int i = 0; i < maskdim; i++) {
                for (int j = 0; j < maskdim; j++) {
                    int cr = row - offset + i;
                    int cc = col - offset + j;
                    if (cr >= 0 && cr < n && cc >= 0 && cc < n) {
                        result[row * n + col] += image[cr * n + cc] * mask[i * maskdim + j];
                    }
                }
            }
        }
    }
}

__global__ void calculate2DConvolution(int *image, int *mask, int *result, int n, int maskdim) {
    int offset = maskdim / 2;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n) {
        int start_row = row - offset;
        int start_col = col - offset;
        for (int i = 0; i < maskdim; i++) {
            for (int j = 0; j < maskdim; j++) {
            if (start_row + i >= 0 && start_row + i < n && start_col + j >= 0 && start_col + j < n) {
                int cr = start_row + i, cc = start_col + j;
                result[row * n + col] += image[cr * n + cc] *
                mask[i * maskdim + j];
                }
            }
        }
    }
}

int main() {
    clock_t start, end;
    start = clock();
    int image[n * n];
    int result[n * n];
    int mask[maskdim * maskdim];
    for (int i = 0; i < n * n; i++) {
        image[i] = rand() % 256; // Randomize the image
    }
    for (int i = 0; i < maskdim * maskdim; i++) {
        mask[i] = rand() % 5; // Randomize the mask
    }
    int *d_image, *d_mask, *d_result;
    
    hipMalloc(&d_image, n * n * sizeof(int));
    hipMalloc(&d_mask, maskdim * maskdim * sizeof(int));
    hipMalloc(&d_result, n * n * sizeof(int));
    hipMemcpy(d_image, image, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, maskdim * maskdim * sizeof(int), hipMemcpyHostToDevice);
    
    int thread = 16;
    int block = (n + thread - 1) / thread;
    
    dim3 blocks(block, block);
    dim3 threads(thread, thread);
    
    // Measure GPU execution time
    hipEvent_t startGPU, stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);
    hipEventRecord(startGPU);
    calculate2DConvolution<<<blocks, threads>>>(d_image, d_mask, d_result, n, maskdim);
    
    hipEventRecord(stopGPU);
    hipEventSynchronize(stopGPU);
    
    float millisecondsGPU = 0;
    hipEventElapsedTime(&millisecondsGPU, startGPU, stopGPU);
    printf("GPU Execution Time: %f ms\n", millisecondsGPU);
    hipMemcpy(result, d_result, n * n * sizeof(int), hipMemcpyDeviceToHost);
    
    // Measure CPU execution time
    start = clock();
    calculate2DConvolutionCPU(image, mask, result, n, maskdim);
    end = clock();
    double durationCPU = ((double)(end - start) * 1000.0) / CLOCKS_PER_SEC;
    
    printf("CPU Execution Time: %f ms\n", durationCPU);
    hipFree(d_image);
    hipFree(d_mask);
    hipFree(d_result);
    return 0;
}