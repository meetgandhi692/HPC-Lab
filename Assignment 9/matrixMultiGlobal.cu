
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
const int n = 2<<8;

__global__ void multiply(int *mat1, int *mat2, int *result, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
        if (row < n && col < n) {
            for (int k = 0; k < n; k++) {
                result[row * n + col] += mat1[row * n + k] * mat2[k * n + col];
        }
    }
}


int main() {
    int mat1[n * n];
    int mat2[n * n];
    for (int i = 0; i < n * n; i++) {
        mat1[i] = i + 1;
        mat2[i] = n * n - i;
    }
    int result[n * n];
    int *d_mat1, *d_mat2, *d_result;
    hipMalloc(&d_mat1, n * n * sizeof(int));
    hipMalloc(&d_mat2, n * n * sizeof(int));
    hipMalloc(&d_result, n * n * sizeof(int));
    hipMemcpy(d_mat1, mat1, n * n * sizeof(int),
    hipMemcpyHostToDevice);
    hipMemcpy(d_mat2, mat2, n * n * sizeof(int),
    hipMemcpyHostToDevice);

    size_t blocks=1;
    size_t threads=64;
    
    multiply<<<blocks, threads>>>(d_mat1, d_mat2, d_result, n);
    hipDeviceSynchronize();
    hipMemcpy(result, d_result, n * n * sizeof(int),
    hipMemcpyDeviceToHost);
    
    return 0;
}