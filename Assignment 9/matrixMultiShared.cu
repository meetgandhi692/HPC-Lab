
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
const int n = 2<<8;

__global__ void multiply(int *mat1, int *mat2, int *result, int n) {
    int row = blockIdx.y;
    int col = blockIdx.x;
    int k = threadIdx.x;
    __shared__ int temp[1000];
    temp[k] = mat1[row * n + k] * mat2[k * n + col];
    __syncthreads();
    for (int i = 0; i < n; i++) {
        result[row * n + col] += temp[i];
    }
}

int main() {
    int mat1[n * n];
    int mat2[n * n];
    for (int i = 0; i < n * n; i++) {
        mat1[i] = i + 1;
        mat2[i] = n * n - i;
    }
    int result[n * n];
    int *d_mat1, *d_mat2, *d_result;
    hipMalloc(&d_mat1, n * n * sizeof(int));
    hipMalloc(&d_mat2, n * n * sizeof(int));
    hipMalloc(&d_result, n * n * sizeof(int));
    hipMemcpy(d_mat1, mat1, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mat2, mat2, n * n * sizeof(int), hipMemcpyHostToDevice);

    size_t blocks=1;
    size_t threads=64;
    
    multiply<<<blocks, threads>>>(d_mat1, d_mat2, d_result, n);

    hipDeviceSynchronize();
    hipMemcpy(result, d_result, n * n * sizeof(int), hipMemcpyDeviceToHost);
    return 0;
}