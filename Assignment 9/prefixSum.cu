
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

// CUDA kernel for parallel prefix sum
__global__ void prefixSumKernel(int *input, int *output, size_t n) {
    extern __shared__ int temp[];
    int tid = threadIdx.x;
    
    // Load data into shared memory
    temp[tid] = input[tid];
    
    __syncthreads();
    for (int offset = 1; offset < n; offset *= 2) {
        int val = 0;
        if (tid >= offset) {
            val = temp[tid - offset];
        }
        __syncthreads();
        temp[tid] += val;
        __syncthreads();
    }
    output[tid] = temp[tid];
}


int main() {
    const size_t arraySize = 2<<24;  // Adjust the size as needed
    const size_t blockSize = 64; // Adjust the block size as needed

    int *hostInput = new int[arraySize];
    for (size_t i = 0; i < arraySize; ++i) {
        hostInput[i] = rand() % 100; // Adjust the range as needed
    }

    int *deviceInput, *deviceOutput;
    hipMalloc((void**)&deviceInput, sizeof(int) * arraySize);
    hipMalloc((void**)&deviceOutput, sizeof(int) * arraySize);

    hipMemcpy(deviceInput, hostInput, sizeof(int) * arraySize, hipMemcpyHostToDevice);

    // Define grid size based on block and array size
    size_t gridSize = (arraySize + blockSize - 1) / blockSize;

    prefixSumKernel<<<gridSize, blockSize, blockSize * sizeof(int)>>>(deviceInput, deviceOutput, arraySize);

    int *hostOutput = new int[arraySize];
    hipMemcpy(hostOutput, deviceOutput, sizeof(int) * arraySize, hipMemcpyDeviceToHost);


    // Free allocated memory
    delete[] hostInput;
    delete[] hostOutput;
    hipFree(deviceInput);
    hipFree(deviceOutput);

    return 0;
}
